/*
other things we should test:
- struct pointer, with offset
- multiple struct pointers, cut from same buffer
- getting values from various types of structs passed in
*/

#include <iostream>
#include <memory>
#include <cassert>

using namespace std;

#include <hip/hip_runtime.h>

struct Struct_fp_fp_f_f {
    float *p1;
    float *p2;
    float f1;
    float f2;
};

struct Struct_fp {
    float *p1;
};

struct Struct_1float {
    float f1;
};

struct Struct_2floats {
    float f1;
    float f2;
};

__global__ void struct_byvalue(struct Struct_fp_fp_f_f mystruct, float *out) {
    out[0] = mystruct.f1;
    out[1] = mystruct.f2;
    mystruct.p1[0] = 9.0f;
    mystruct.p2[0] = 10.0f;
}

void testbyvaluestruct() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *gpuFloats1;
    hipMalloc((void**)(&gpuFloats1), N * sizeof(float));

    float *gpuFloats2;
    hipMalloc((void**)(&gpuFloats2), N * sizeof(float));

    float *gpuFloats3;
    hipMalloc((void**)(&gpuFloats3), N * sizeof(float));

    float *gpuOut;
    hipMalloc((void**)(&gpuOut), N * sizeof(float));

    float *hostFloats1 = new float[N];
    float *hostFloats2 = new float[N];
    float *hostFloats3 = new float[N];
    float *hostOut = new float[N];

    struct Struct_fp_fp_f_f mystruct = {(float *)gpuFloats1, (float *)gpuFloats2, 3.0f, 8.0f};
    struct_byvalue<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, (float *)gpuOut);

    hipMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostFloats2, gpuFloats2, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostOut, gpuOut, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;
    cout << hostFloats2[0] << endl;
    cout << hostOut[0] << endl;
    cout << hostOut[1] << endl;

    assert(hostFloats1[0] == 9);
    assert(hostFloats2[0] == 10);
    assert(hostOut[0] == 3);
    assert(hostOut[1] == 8);

    hipFree(gpuFloats1);
    hipFree(gpuFloats2);
    hipFree(gpuFloats3);
    hipFree(gpuOut);

    delete[]hostFloats1;
    delete[]hostFloats2;
    delete[]hostFloats3;
    delete[]hostOut;

    hipStreamDestroy(stream);
}

__global__ void struct_aspointer(struct Struct_2floats *mystruct, float *out) {
    out[0] = mystruct->f1;
    out[1] = mystruct->f2;
}

void testaspointerstruct() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostOut = new float[N];

    float *gpuOut;
    hipMalloc((void**)(&gpuOut), N * sizeof(float));

    struct Struct_2floats mystruct = { 5, 7 };
    struct Struct_2floats *gpu_mystruct;
    hipMalloc((void**)(&gpu_mystruct), sizeof(mystruct));
    hipMemcpy(gpu_mystruct, &mystruct, sizeof(mystruct), hipMemcpyHostToDevice);

    struct_aspointer<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(gpu_mystruct, gpuOut);

    hipMemcpy(hostOut, gpuOut, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    cout << hostOut[0] << endl;
    cout << hostOut[1] << endl;

    assert(hostOut[0] == 5);
    assert(hostOut[1] == 7);

    delete[]hostOut;

    hipStreamDestroy(stream);
}

__global__ void kernel_twostructs(struct Struct_fp_fp_f_f mystruct, struct Struct_fp mystruct2) {
    mystruct.p1[0] = 9.0f;
    mystruct.p2[0] = 10.0f;
    mystruct2.p1[0] = 11.0f;
}

void testtwostructs() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *gpuFloats1;
    hipMalloc((void**)(&gpuFloats1), N * sizeof(float));

    float *gpuFloats2;
    hipMalloc((void**)(&gpuFloats2), N * sizeof(float));

    float *gpuFloats3;
    hipMalloc((void**)(&gpuFloats3), N * sizeof(float));

    float *hostFloats1 = new float[N];
    float *hostFloats2 = new float[N];
    float *hostFloats3 = new float[N];

    struct Struct_fp_fp_f_f mystruct = {(float *)gpuFloats1, (float *)gpuFloats2};
    struct Struct_fp mystruct2 = {(float *)gpuFloats3};

    kernel_twostructs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct, mystruct2);

    hipMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostFloats2, gpuFloats2, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hostFloats3, gpuFloats3, 4 * sizeof(float), hipMemcpyDeviceToHost);

    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;
    cout << hostFloats2[0] << endl;
    cout << hostFloats3[0] << endl;

    assert(hostFloats1[0] == 9);
    assert(hostFloats2[0] == 10);
    assert(hostFloats3[0] == 11);

    hipFree(gpuFloats1);
    hipFree(gpuFloats2);
    hipFree(gpuFloats3);

    delete[]hostFloats1;
    delete[]hostFloats2;
    delete[]hostFloats3;

    hipStreamDestroy(stream);
}

// __global__ void kernel_structbyval_noptrs(struct Struct_1float mystruct1, float *out) {
//     if(threadIdx.x == 0) {
//         out[0] = mystruct1.p1;
//         out[1] = 5;
//     }
// }

// void teststructbyvalNoPtr() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostFloats1 = new float[N];

//     float *gpuFloats1;
//     cudaMalloc((void**)(&gpuFloats1), N * sizeof(float));

//     struct Struct_1float mystruct1 = {8.0f};

//     kernel_structbyval_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct1, (float *)gpuFloats1);

//     cudaMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), cudaMemcpyDeviceToHost);
//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats1[1] << endl;

//     assert(hostFloats1[0] == 8);

//     delete[] hostFloats1;
//     cudaFree(gpuFloats1);

//     cuStreamDestroy(stream);
// }

// __global__ void kernel_struct2byval_noptrs(struct Struct_fpNoPtr mystruct1, float *out) {
//     if(threadIdx.x == 0) {
//     out[0] = mystruct1.p1;
//     out[1] = mystruct1.p2;
//     }
// }

// void teststruct2byvalNoPtr() {
//     int N = 1024;

//     CUstream stream;
//     cuStreamCreate(&stream, 0);

//     float *hostFloats1;
//     cuMemHostAlloc((void **)&hostFloats1, N * sizeof(float), CU_MEMHOSTALLOC_PORTABLE);

//     CUdeviceptr gpuFloats1;
//     cuMemAlloc(&gpuFloats1, N * sizeof(float));
//     cuMemcpyHtoDAsync((CUdeviceptr)(((float *)gpuFloats1)), hostFloats1, N * sizeof(float), stream);

//     struct Struct_fpNoPtr mystruct1 = {8.0f, 9.0f};

//     kernel_struct2byval_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(mystruct1, (float *)gpuFloats1);
//     cuMemcpyDtoHAsync(hostFloats1, gpuFloats1, N * sizeof(float), stream);
//     cuStreamSynchronize(stream);

//     cout << hostFloats1[0] << endl;
//     cout << hostFloats1[1] << endl;

//     assert(hostFloats1[0] == 8);
//     assert(hostFloats1[1] == 9);

//     cuMemFreeHost(hostFloats1);
//     cuMemFree(gpuFloats1);

//     cuStreamDestroy(stream);
// }

__global__ void kernel_twostructs_noptrs(struct Struct_2floats *mystruct, struct Struct_1float *mystruct2, struct Struct_1float mystruct3, float *out) {
    if(threadIdx.x == 0) {
    out[0] = mystruct->f1;
    out[1] = mystruct->f2;
    out[2] = mystruct2->f1;
    out[3] = mystruct3.f1;
    }
}

void test_twostructs_byptr_NoPtr() {
    int N = 1024;

    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, 0);

    float *hostFloats1 = new float[N];

    float *gpuFloats1;
    hipMalloc((void**)(&gpuFloats1), N * sizeof(float));

    struct Struct_2floats mystruct = {5.0f, 6.0f};
    struct Struct_1float mystruct2 = {7.0f};
    struct Struct_1float mystruct3 = {8.0f};

    struct Struct_2floats *gpu_mystruct;
    hipMalloc((void**)(&gpu_mystruct), sizeof(mystruct));
    hipMemcpy(gpu_mystruct, &mystruct, sizeof(mystruct), hipMemcpyHostToDevice);

    struct Struct_1float *gpu_mystruct2;
    hipMalloc((void**)(&gpu_mystruct2), sizeof(mystruct2));
    hipMemcpy(gpu_mystruct2, &mystruct2, sizeof(mystruct2), hipMemcpyHostToDevice);

    kernel_twostructs_noptrs<<<dim3(1,1,1), dim3(32,1,1), 0, stream>>>(gpu_mystruct, gpu_mystruct2, mystruct3, (float *)gpuFloats1);
    hipMemcpy(hostFloats1, gpuFloats1, 4 * sizeof(float), hipMemcpyDeviceToHost);
    hipStreamSynchronize(stream);

    cout << hostFloats1[0] << endl;
    cout << hostFloats1[1] << endl;
    cout << hostFloats1[2] << endl;
    cout << hostFloats1[3] << endl;

    assert(hostFloats1[0] == 5);
    assert(hostFloats1[1] == 6);
    assert(hostFloats1[2] == 7);
    assert(hostFloats1[3] == 8);

    hipFree(gpuFloats1);
    hipFree(gpu_mystruct);
    hipFree(gpu_mystruct2);
    // cudaFree(gpu_mystruct3);

    delete[] hostFloats1;

    hipStreamDestroy(stream);
}

int main(int argc, char *argv[]) {
    cout << "\ntestvaluestruct" << endl;
    testbyvaluestruct();

    cout << "\ntestaspointersstruct" << endl;
    testaspointerstruct();

    cout << "\ntesttwostructs" << endl;
    testtwostructs();

    // cout << "\teststruct2byvalNoPtr" << endl;
    // teststruct2byvalNoPtr();

    // cout << "\teststructbyvalNoPtr" << endl;
    // teststructbyvalNoPtr();

    cout << "\ntest_twostructs_byptr_NoPtr" << endl;
    test_twostructs_byptr_NoPtr();

    return 0;
}